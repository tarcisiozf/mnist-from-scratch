#include "hip/hip_runtime.h"
#include "cuda.cuh"

#include <stdlib.h>
#include <stdio.h>
#include <memory.h>
#include <math.h>

Matrix* matrix_create(int rows, int cols) {
    Matrix* m = (Matrix*) malloc(sizeof(Matrix));
    m->rows = rows;
    m->cols = cols;
    m->data = (double*) malloc(rows * cols * sizeof(double));
    memset(m->data, 0, rows * cols * sizeof(double));
    return m;
}

Matrix* matrix_from_shape(Matrix* m) {
    return matrix_create(m->rows, m->cols);
}

Matrix* matrix_from_data(int rows, int cols, double* data) {
    Matrix* m = (Matrix*) malloc(sizeof(Matrix));
    m->rows = rows;
    m->cols = cols;
    m->data = data;
    return m;
}

void matrix_free(Matrix* m) {
    if (m == NULL) {
        return;
    }
    free(m->data);
    free(m);
}

__global__ void matrixMultiplyKernel(double *A, double *B, double *C, int N, int M, int K) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < N && col < K) {
        double value = 0;

        for (int i = 0; i < M; ++i) {
            value += A[row * M + i] * B[i * K + col];
        }

        C[row * K + col] = value;
    }
}

Matrix* matrix_dot(Matrix* a, Matrix* b) {
    if (a->cols != b->rows) {
        printf("Error: Failed to multiply shapes (%d, %d) and (%d, %d)\n", a->rows, a->cols, b->rows, b->cols);
        exit(1);
    }

    Matrix* c = matrix_create(a->rows, b->cols);

    double *d_A, *d_B, *d_C;

    size_t size_A = a->rows * a->cols * sizeof(double);
    size_t size_B = b->rows * b->cols * sizeof(double);
    size_t size_C = a->rows * b->cols * sizeof(double);

    hipMalloc((void**)&d_A, size_A);
    hipMalloc((void**)&d_B, size_B);
    hipMalloc((void**)&d_C, size_C);

    hipMemcpy(d_A, a->data, size_A, hipMemcpyHostToDevice);
    hipMemcpy(d_B, b->data, size_B, hipMemcpyHostToDevice);

    dim3 dimBlock(16, 16);
    dim3 dimGrid((b->cols + dimBlock.x - 1) / dimBlock.x, (a->rows + dimBlock.y - 1) / dimBlock.y);

    matrixMultiplyKernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_C, a->rows, a->cols, b->cols);

    hipMemcpy(c->data, d_C, size_C, hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return c;
}

Matrix* matrix_transpose(Matrix* m) {
    Matrix* t = matrix_create(m->cols, m->rows);
    for (int i = 0; i < m->rows; i++) {
        for (int j = 0; j < m->cols; j++) {
            t->data[j * t->cols + i] = m->data[i * m->cols + j];
        }
    }
    return t;
}

Matrix* matrix_broadcast(Matrix* a, Matrix* b) {
    if (a->rows == b->rows && a->cols > b->cols) {
        Matrix* c = matrix_create(b->rows, a->cols);
        for (int i = 0; i < b->rows; i++) {
            for (int j = 0; j < a->cols; j++) {
                c->data[i * c->cols + j] = b->data[i * b->cols];
            }
        }
        return c;
    }
    if (a->rows > b->rows && a->cols == b->cols) {
        Matrix* c = matrix_create(a->rows, b->cols);
        for (int i = 0; i < a->rows; i++) {
            for (int j = 0; j < b->cols; j++) {
                c->data[i * c->cols + j] = b->data[j];
            }
        }
        return c;
    }
    printf("Error: Failed to broadcast shapes (%d, %d) and (%d, %d)\n", a->rows, a->cols, b->rows, b->cols);
    exit(1);
}

Matrix* matrix_add(Matrix* a, Matrix* b) {
    char did_broadcast = 0;
    if (a->rows != b->rows || a->cols != b->cols) {
        b = matrix_broadcast(a, b);
        did_broadcast = 1;
    }

    Matrix* c = matrix_create(a->rows, a->cols);
    for (int i = 0; i < a->rows; i++) {
        for (int j = 0; j < a->cols; j++) {
            c->data[i * c->cols + j] = a->data[i * a->cols + j] + b->data[i * b->cols + j];
        }
    }

    if (did_broadcast) {
        matrix_free(b);
    }

    return c;
}

Matrix* matrix_sub(Matrix* a, Matrix* b) {
    if (a->rows != b->rows || a->cols != b->cols) {
        printf("Error: Failed to subtract shapes (%d, %d) and (%d, %d)\n", a->rows, a->cols, b->rows, b->cols);
        exit(1);
    }

    Matrix* c = matrix_from_shape(a);
    for (int i = 0; i < a->rows; i++) {
        for (int j = 0; j < a->cols; j++) {
            c->data[i * c->cols + j] = a->data[i * a->cols + j] - b->data[i * b->cols + j];
        }
    }
    return c;
}

Matrix* matrix_mul(Matrix* a, Matrix* b) {
    if (a->rows != b->rows || a->cols != b->cols) {
        printf("Error: Failed to multiply shapes (%d, %d) and (%d, %d)\n", a->rows, a->cols, b->rows, b->cols);
        exit(1);
    }

    Matrix* c = matrix_from_shape(a);
    for (int i = 0; i < a->rows; i++) {
        for (int j = 0; j < a->cols; j++) {
            c->data[i * c->cols + j] = a->data[i * a->cols + j] * b->data[i * b->cols + j];
        }
    }
    return c;
}

Matrix *matrix_div(Matrix *a, Matrix *b) {
    char did_broadcast = 0;
    if (a->rows != b->rows || a->cols != b->cols) {
        b = matrix_broadcast(a, b);
        did_broadcast = 1;
    }

    Matrix* c = matrix_from_shape(a);
    for (int i = 0; i < a->rows; i++) {
        for (int j = 0; j < a->cols; j++) {
            c->data[i * c->cols + j] = a->data[i * a->cols + j] / b->data[i * b->cols + j];
        }
    }

    if (did_broadcast) {
        matrix_free(b);
    }

    return c;
}

Matrix* matrix_divf(Matrix* a, double f) {
    Matrix* c = matrix_from_shape(a);
    for (int i = 0; i < a->rows; i++) {
        for (int j = 0; j < a->cols; j++) {
            c->data[i * c->cols + j] = a->data[i * a->cols + j] / f;
        }
    }
    return c;
}

Matrix* matrix_subf(Matrix* m, double f) {
    Matrix* c = matrix_from_shape(m);
    for (int i = 0; i < m->rows; i++) {
        for (int j = 0; j < m->cols; j++) {
            c->data[i * c->cols + j] = m->data[i * m->cols + j] - f;
        }
    }
    return c;
}

Matrix* matrix_mulf(Matrix* m, double f) {
    Matrix* c = matrix_from_shape(m);
    for (int i = 0; i < m->rows; i++) {
        for (int j = 0; j < m->cols; j++) {
            c->data[i * c->cols + j] = m->data[i * m->cols + j] * f;
        }
    }
    return c;
}

void matrix_print(char* label, Matrix* m, int y, int x) {
    printf("%s\n", label);
    int rows;
    int cols;

    if (y == -1 && x == -1) {
        rows = m->rows;
        cols = m->cols;
    } else {
        rows = y;
        cols = x;
    }
    if (rows > m->rows) {
        rows = m->rows;
    }
    if (cols > m->cols) {
        cols = m->cols;
    }

    for (int i = 0; i < rows; i++) {
        for (int j = 0; j < cols; j++) {
            printf("%f ", m->data[i * m->cols + j]);
        }
        printf("\n");
    }
    printf("--------------\n");
}

// Uses Kahan summation for better precision
double matrix_sum(Matrix* m) {
    double sum = 0.0;
    double c = 0.0;
    for (int i = 0; i < m->rows; i++) {
        for (int j = 0; j < m->cols; j++) {
            double y = m->data[i * m->cols + j] - c;
            double t = sum + y;
            c = (t - sum) - y;
            sum = t;
        }
    }
    return sum;
}

Matrix* matrix_rand(int rows, int cols) {
    Matrix* m = matrix_create(rows, cols);
    for (int i = 0; i < rows * cols; i++) {
        m->data[i] = (double) rand() / RAND_MAX - 0.5;
    }
    return m;
}

Matrix* matrix_relu(Matrix* m) {
    Matrix* c = matrix_from_shape(m);
    int idx;
    double val;
    for (int i = 0; i < m->rows; i++) {
        for (int j = 0; j < m->cols; j++) {
            idx = i * m->cols + j;
            val = m->data[idx];
            c->data[idx] = val > 0 ? val : 0;
        }
    }
    return c;
}

Matrix* matrix_softmax(Matrix* m) {
    Matrix* c = matrix_from_shape(m);
    Matrix* sum = matrix_create(1, m->cols);
    for (int i = 0; i < c->rows; i++) {
        for (int j = 0; j < c->cols; j++) {
            c->data[i * c->cols + j] = exp(m->data[i * m->cols + j]);
            sum->data[j] += c->data[i * c->cols + j];
        }
    }
    Matrix* out = matrix_div(c, sum);
    matrix_free(c);
    matrix_free(sum);
    return out;
}

Matrix* matrix_one_hot(const double* Y, int len) {
    Matrix* m = matrix_create(len, 10); // int(max)+1
    for (int i = 0; i < len; i++) {
        m->data[i * 10 + (int) Y[i]] = 1;
    }
    Matrix* out = matrix_transpose(m);
    matrix_free(m);
    return out;
}

Matrix* matrix_cols(Matrix* m, int start, int end) {
    Matrix* c = matrix_create(m->rows, end - start);
    for (int i = 0; i < m->rows; i++) {
        for (int j = start; j < end; j++) {
            c->data[i * c->cols + j - start] = m->data[i * m->cols + j];
        }
    }
    return c;
}